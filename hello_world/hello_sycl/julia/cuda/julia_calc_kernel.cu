
#include <hip/hip_runtime.h>
#include <stdint.h>
#include <stdio.h>

// NOTE: 这里误指定 zx, zy 为 int 竟然能编译通过...导致结果错误
__device__ int HowManySteps(float zx, float zy, float cx, float cy) {
  float zx2 = 0.0;
  float zy2 = 0.0;
  float abs_sq = 0.0;

  int MAX_ITERS = 255;
  float DIVERGENCE_LIMIT = 2.0;

  for (size_t i = MAX_ITERS; i > 0; i--) {
    zx2 = zx * zx - zy * zy + cx;
    zy2 = 2.0 * zx * zy + cy;

    zx = zx2;
    zy = zy2;

    abs_sq = zx * zx + zy * zy;

    if (abs_sq >= DIVERGENCE_LIMIT) {
      return i;
    }
  }

  return 0;
}

__global__ void JuliaKernel(
    int height, int width, int zoom, uchar4 *dev_data, float cx, float cy) {
  int x = threadIdx.x;
  int y = blockIdx.x;

  float zx = (x - 0.5 * width) / (0.5 * width * zoom);
  float zy = (y - 0.5 * height) / (0.5 * height * zoom);

  int count = HowManySteps(zx, zy, cx, cy);
  int color = (count << 21) + (count << 10) + (count << 3);
  dev_data[x * height + y] = {
      (uint8_t)(color >> 16), (uint8_t)(color >> 8), (uint8_t)color,
      (uint8_t)255};
}

void Julia(int height, int width, int zoom, void *data, float cx, float cy) {
  static uchar4 *dev_data = 0;
  if (dev_data == 0) {
    hipMalloc(&dev_data, sizeof(uchar4) * height * width);
  }
  JuliaKernel<<<height, width>>>(height, width, zoom, dev_data, cx, cy);
  // cudaError_t err = cudaGetLastError();
  // if (err != cudaSuccess) {
  //   printf("CUDA Error: %s\n", cudaGetErrorString(err));
  // }
  hipDeviceSynchronize();
  hipMemcpy(
      data, dev_data, sizeof(uchar4) * height * width, hipMemcpyDeviceToHost);
}
