
#include <hip/hip_runtime.h>
#include <stdint.h>

__device__ int HowManySteps(int zx, int zy) {
  float cx = 0.285;
  float cy = 0.01;

  float zx2 = 0.0;
  float zy2 = 0.0;
  float abs_sq = 0.0;

  int MAX_ITERS = 255;
  float DIVERGENCE_LIMIT = 2.0;

  for (size_t i = MAX_ITERS; i > 0; i--) {
    zx2 = zx * zx - zy * zy + cx;
    zy2 = 2.0 * zx * zy + cy;

    zx = zx2;
    zy = zy2;

    abs_sq = zx * zx + zy * zy;

    if (abs_sq >= DIVERGENCE_LIMIT) {
      return i;
    }
  }

  return 0;
}

__global__ void JuliaKernel(int height, int width, int zoom, uchar4 *dev_data) {
  int x = threadIdx.x;
  int y = blockIdx.x;

  float zx = (x - 0.5 * width) / (0.5 * width * zoom);
  float zy = (y - 0.5 * height) / (0.5 * height * zoom);

  int count = HowManySteps(zx, zy);
  int color = (count << 21) + (count << 10) + (count << 3);
  dev_data[x * height + y] = {
      (uint8_t)(color >> 16), (uint8_t)(color >> 8), (uint8_t)color,
      (uint8_t)255};
}

void Julia(int height, int width, int zoom, void *data) {
  uchar4 *dev_data;
  hipMalloc(&dev_data, sizeof(uchar4) * height * width);
  JuliaKernel<<<height, width>>>(height, width, zoom, dev_data);
  hipMemcpy(
      data, dev_data, sizeof(uchar4) * height * width, hipMemcpyDeviceToHost);
}
