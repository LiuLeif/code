
#include <hip/hip_runtime.h>
#include <stdint.h>
#include <stdio.h>

// NOTE: 这里误指定 zx, zy 为 int 也能编译通过...导致结果错误
__device__ int HowManySteps(float zx, float zy, float cx, float cy) {
  float zx2 = 0.0;
  float zy2 = 0.0;
  float norm = 0.0;

  int MAX_ITERS = 255;
  float DIVERGENCE_LIMIT = 2.0;

  for (size_t i = MAX_ITERS; i > 0; i--) {
    zx2 = zx * zx - zy * zy + cx;
    zy2 = 2.0 * zx * zy + cy;

    zx = zx2;
    zy = zy2;

    norm = zx * zx + zy * zy;

    if (norm >= DIVERGENCE_LIMIT) {
      return i;
    }
  }

  return 0;
}

__global__ void JuliaKernel(
    int height, int width, float zoom, uchar4 *dev_data, float cx, float cy,
    float center_x, float center_y) {
  int global_id = blockIdx.x * blockDim.x + threadIdx.x;
  int x = (int)(global_id / height);
  int y = global_id - x * height;

  float zx = (x - 0.5 * width) / (0.5 * width * zoom) + center_x;
  float zy = (y - 0.5 * height) / (0.5 * height * zoom) + center_y;

  int count = HowManySteps(zx, zy, cx, cy);
  int color = (count << 21) + (count << 10) + (count << 3);
  dev_data[x * height + y] = {
      (uint8_t)(color >> 16), (uint8_t)(color >> 8), (uint8_t)color,
      (uint8_t)255};
}

void Julia(
    int height, int width, float zoom, void *data, float cx, float cy,
    float center_x, float center_y) {
  static uchar4 *dev_data = 0;
  if (dev_data == 0) {
    hipMalloc(&dev_data, sizeof(uchar4) * height * width);
  }
  // NOTE: 直接指定 kernel shape 为 (height, width) 不可行, 因为一个 block最多只
  // 能有 1024 个 thread, 导致 width 不能超过 1024, 这里是模拟了sycl 的 range 方
  // 法
  JuliaKernel<<<ceil((height * width) / 32), 32>>>(
      height, width, zoom, dev_data, cx, cy, center_x, center_y);
  hipDeviceSynchronize();
  hipMemcpy(
      data, dev_data, sizeof(uchar4) * height * width, hipMemcpyDeviceToHost);
}
