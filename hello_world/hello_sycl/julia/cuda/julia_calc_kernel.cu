
#include <hip/hip_runtime.h>
#include <stdint.h>
#include <stdio.h>

// NOTE: 这里误指定 zx, zy 为 int 也能编译通过...导致结果错误
__device__ int HowManySteps(float zx, float zy, float cx, float cy) {
  float zx2 = 0.0;
  float zy2 = 0.0;
  float abs_sq = 0.0;

  int MAX_ITERS = 255;
  float DIVERGENCE_LIMIT = 2.0;

  for (size_t i = MAX_ITERS; i > 0; i--) {
    zx2 = zx * zx - zy * zy + cx;
    zy2 = 2.0 * zx * zy + cy;

    zx = zx2;
    zy = zy2;

    abs_sq = zx * zx + zy * zy;

    if (abs_sq >= DIVERGENCE_LIMIT) {
      return i;
    }
  }

  return 0;
}

__global__ void JuliaKernel(
    int height, int width, float zoom, uchar4 *dev_data, float cx, float cy,
    float center_x, float center_y) {
  int global_id = blockIdx.x * blockDim.x + threadIdx.x;
  int x = (int)(global_id / height);
  int y = global_id - x * height;

  float zx = (x - 0.5 * width) / (0.5 * width * zoom) + center_x;
  float zy = (y - 0.5 * height) / (0.5 * height * zoom) + center_y;

  int count = HowManySteps(zx, zy, cx, cy);
  int color = (count << 21) + (count << 10) + (count << 3);
  dev_data[x * height + y] = {
      (uint8_t)(color >> 16), (uint8_t)(color >> 8), (uint8_t)color,
      (uint8_t)255};
}

void Julia(
    int height, int width, float zoom, void *data, float cx, float cy,
    float center_x, float center_y) {
  static uchar4 *dev_data = 0;
  if (dev_data == 0) {
    hipMalloc(&dev_data, sizeof(uchar4) * height * width);
  }
  // NOTE: 直接指定 kernel shape 为 (height, width) 不可行, 因为一个 block
  // 最多只能有 1024 个 thread, 导致 width 不能超过 1024, 这里实现上是模拟了
  // sycl 的 range 方法
  JuliaKernel<<<ceil((height * width) / 32), 32>>>(
      height, width, zoom, dev_data, cx, cy, center_x, center_y);
  // cudaError_t err = cudaGetLastError();
  // if (err != cudaSuccess) {
  //   printf("CUDA Error: %s\n", cudaGetErrorString(err));
  // }
  hipDeviceSynchronize();
  hipMemcpy(
      data, dev_data, sizeof(uchar4) * height * width, hipMemcpyDeviceToHost);
}
